#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
 
struct Pixel
{
 unsigned char r, g, b, a;
};
 
void ConvertImageToGrayCpu(unsigned char* imageRGBA, int width, int height)
{
    const int Ykernel = 3;
    const int Xkernel = 3;
    int sum1 = 0;
    int sum2 = 0;
    int sum3 = 0;
    float kernel[Ykernel][Xkernel] =
            { 
            {0, -1, 0},
            {-1, 8, -1},
            {0, -1, 0}
            };


    for (int y = 0; y < height - 2; y++)
    {
        for (int x = 0; x < width - 2; x++)
        {
            for(int i = 0; i < Ykernel; i++)
            {
                for(int j = 0; j <Xkernel; j++)
                {
                    Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
                    char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
                    sum1 += (pixelValue * kernel[i][j]);
                    sum2 += (pixelValue * kernel[i][j]);
                    sum3 += (pixelValue * kernel[i][j]);
                    //printf("sum1 = %d\n ",sum1);
                }
            }
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            ptrPixel->r = sum1;
            ptrPixel->g = sum2;
            ptrPixel->b = sum3;
            ptrPixel->a = 255;

            sum1 = 0;
            sum2 = 0;
            sum3 = 0;            
        }
    }
}
 /*
__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA, int width, int height, unsigned char *NewImage )
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(y < height && x < width)
    {
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r/3 + ptrPixel->g / 3 + ptrPixel->b / 3);
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;
    }

}


  if(y < height && x < width)
    {  

         
            const int Xkernel = 3;
            const int Ykernel = 3;

            int kernel[Ykernel][Xkernel] =
            { 
                {1,0,-1},
                {1,0,-1},
                {1,0,-1}
            };

        for(int i = 0; i < Ykernel; i++)
            {
                int value = 0;
                for(int j = 0; j < Xkernel; j++)
                {  
                    int x_offset = x + i - 3/2;
                    int y_offset = y + j - 3/2; 
                    
                    if(x_offset >= 0 && x_offset < width && y_offset >= 0 && y_offset < height)
                    {
                        Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
                        unsigned char pixelValue = (unsigned char)(ptrPixel->r ptrPixel->g + ptrPixel->b);
                        int value = pixelValue;
                    }
                    NewImage = value * kernel[i][j];                
                    
                }
            } 

    }



}
*/

int main(int argc, char** argv)
{

    // Check argument count
    if (argc < 2)
    {
        printf("Usage: im2gray <filename>\r\n");
        return -1;
    }
 
    // Open image
    int width, height, componentCount;
    printf("Loading png file...\r\n");
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    unsigned char* OutputImage; 
    if (!imageData)
    {
        printf("Failed to open Image\r\n");
        return -1;
    }
    printf(" DONE \r\n" );
 
 
    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        printf("Width and/or Height is not dividable by 32!\r\n");
        return -1;
    }
 
    
    // Process image on cpu
    printf("Processing image...\r\n");
    ConvertImageToGrayCpu(imageData, width, height);
    printf(" DONE \r\n");
    
    // Copy data to the gpu
    printf("Copy data to GPU...\r\n");
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    printf(" DONE \r\n");
  
    // Process image on gpu
    printf("Running CUDA Kernel...\r\n");
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
  //  ConvertImageToGrayGpu<<<gridSize, blockSize>>>(ptrImageDataGpu, height, width, OutputImage);
    printf(" DONE \r\n" ); 
 
    // Copy data from the gpu
    printf("Copy data from GPU...\r\n");
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    printf(" DONE \r\n");
 
    // Build output filename
    const char * fileNameOut= "test.png";

    // Write image back to disk
    printf("Writing png to disk...\r\n");
    stbi_write_png(fileNameOut, width, height, 4, imageData, 4 * width);
    printf("DONE\r\n");
 
    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
    
}