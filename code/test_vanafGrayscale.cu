#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
 
struct Pixel
{
 unsigned char r, g, b, a;
};
 
void ConvertImageToGrayCpu(unsigned char* imageRGBA, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;
        }
    }
}
 
__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA, int width, int height )
{
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  


  if(y < height && x < width)
    {   
            int Xkernel = 3;
            int Ykernel = 3;

            int kernel[Ykernel][Xkernel] =
            { 
                {1,0,-1},
                {1,0,-1},
                {1,0,-1}
            };

        for(int i = 0; i < Ykernel; i++)
            {
                for(int j = 0; j < Xkernel; j++)
                {  
                    int x_offset = x + i - 3/2;
                    int y_offset = y + j - 3/2; 
                    
                    
                    
                    
                    
                    
                    
                     /*
                    Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
                    unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
                    ptrPixel->r = pixelValue;
                    ptrPixel->g = pixelValue;
                    ptrPixel->b = pixelValue;
                    ptrPixel->a = 255; 
                    printf("test\r\n");
                    */
     
                }
            } 
           
    }

}
  
int main(int argc, char** argv)
{
    for(int i = 1; i < argc; i++)
    {
    // Check argument count
    if (argc < 2)
    {
        printf("Usage: im2gray <filename>\r\n");
        return -1;
    }
 
    // Open image
    int width, height, componentCount;
    printf("Loading png file...\r\n");
    unsigned char* imageData = stbi_load(argv[i], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        printf("Failed to open Image\r\n");
        return -1;
    }
    printf(" DONE \r\n" );
 
 
    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        printf("Width and/or Height is not dividable by 32!\r\n");
        return -1;
    }
 
    
    // Process image on cpu
    printf("Processing image...\r\n");
    ConvertImageToGrayCpu(imageData, width, height);
    printf(" DONE \r\n");
    
    // Copy data to the gpu
    printf("Copy data to GPU...\r\n");
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    printf(" DONE \r\n");
 
    // Process image on gpu
    printf("Running CUDA Kernel...\r\n");
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    ConvertImageToGrayGpu<<<gridSize, blockSize>>>(ptrImageDataGpu, height, width);
    printf(" DONE \r\n" ); 
 
    // Copy data from the gpu
    printf("Copy data from GPU...\r\n");
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    printf(" DONE \r\n");
 
    // Build output filename
    const char * fileNameOut;
    if(i == 1)
    {
       fileNameOut = "a4.png";
    }
    else if(i == 2)
    {
       
        fileNameOut = "aaaaa.png";
    }
    // Write image back to disk
    printf("Writing png to disk...\r\n");
    stbi_write_png(fileNameOut, width, height, 4, imageData, 4 * width);
    printf("DONE\r\n");
 
    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
    }
} 